#include "hip/hip_runtime.h"
#include <stdio.h>
#include <dirent.h>
#include <stdlib.h>
#include <png.h>
#include <math.h>
#include <string.h>
#include "orig.c"
#include "typedefs.c"

int threads = 1;
int blocks = 1;
int PALETTE_SIZE = 16;

__device__ int* cisSortedCorrectly(color* a, int channel, int length) {
    int isCorrect = true;
    int* correctness = (int*)malloc(sizeof(int) * length - 1);
    for (int i = 0; i < length - 1; i++) {
        if (a[i].values[channel] > a[i + 1].values[channel]) {
            correctness[i] = false;
            isCorrect = false;
            continue;
        }
        correctness[i] = true;
    }
    correctness[length - 1] = isCorrect;
    return correctness;
}


__device__ void mergeChunk(int channel, color* firstChunk, int firstLength, color* secondChunk, int secondLength) {
    color* tempArrayLeft = (color*)malloc(sizeof(*tempArrayLeft) * (firstLength));
    for (int i = 0; i < firstLength; i++) {
        tempArrayLeft[i] = firstChunk[i];
    }
    color* tempArrayRight = (color*)malloc(sizeof(*tempArrayRight) * (secondLength));
    for (int i = 0; i < secondLength; i++) {
        tempArrayRight[i] = secondChunk[i];
    }
    int i = 0, x = 0, y = 0;
    while (i < firstLength + secondLength) {
        if (x == firstLength) {
            firstChunk[i] = tempArrayRight[y];
            y++;
        }
        else if (y == secondLength) {
            firstChunk[i] = tempArrayLeft[x];
            x++;
        }
        else {
            if (tempArrayRight[y].values[channel] < tempArrayLeft[x].values[channel]) {
                firstChunk[i] = tempArrayRight[y];
                y++;
            }
            else {
                firstChunk[i] = tempArrayLeft[x];
                x++;
            }
        }
        i++;
    }
    free(tempArrayLeft);
    free(tempArrayRight);
}

__device__ void mergeSortChunk(int channel, color* bucket, int length) {
    if (length == 1) {
        return;
    }
    else if (length == 2) {
        color px1 = bucket[0], px2 = bucket[1];
        if (px1.values[channel] > px2.values[channel]) {
            bucket[1] = px1;
            bucket[0] = px2;
        }
        return;
    }
    else {
        int mid = length / 2;
        mergeSortChunk(channel, bucket, mid);
        mergeSortChunk(channel, bucket + mid, length - mid);
        color* tempArrayLeft = (color*)malloc(sizeof(*tempArrayLeft) * (mid));
        for (int i = 0; i < mid; i++) {
            tempArrayLeft[i] = bucket[i];
        }
        color* tempArrayRight = (color*)malloc(sizeof(*tempArrayRight) * (length - mid));
        for (int i = 0; i < length - mid; i++) {
            tempArrayRight[i] = bucket[i + mid];
        }
        int i = 0, x = 0, y = 0;
        while (i < length) {
            if (x == mid) {
                bucket[i] = tempArrayRight[y];
                y++;
            }
            else if (y == length - mid) {
                bucket[i] = tempArrayLeft[x];
                x++;
            }
            else {
                if (tempArrayRight[y].values[channel] < tempArrayLeft[x].values[channel]) {
                    bucket[i] = tempArrayRight[y];
                    y++;
                }
                else {
                    bucket[i] = tempArrayLeft[x];
                    x++;
                }
            }
            i++;
        }
        free(tempArrayLeft);
        free(tempArrayRight);
    }
}

__device__ int getChunkLength(int firstChunk, int lastChunk, int threadAmount, int bucketLength) {
    int base = bucketLength / threadAmount;
    int remainder = bucketLength % threadAmount;
    if (lastChunk <= remainder) {
        return (lastChunk - firstChunk) * (base + 1);
    }
    if (firstChunk < remainder) {
        return (lastChunk - remainder) * base + (remainder - firstChunk) * (base + 1);
    }
    return (lastChunk - firstChunk) * base;
}

__global__ void sortBucket(int channel, color* bucket, int length) {
    // Control variable starts at the thread's id and is halved after each iteration.
    int threadAmount = gridDim.x * blockDim.x;
    int iterationControl = blockDim.x * blockIdx.x + threadIdx.x; 
    int threadIndex = iterationControl;
    if (threadIndex >= length) {
        return;
    }
    int range = 1;
    int chunkStart = getChunkLength(0, threadIndex, threadAmount, length);
    int chunkLength = getChunkLength(threadIndex, threadIndex + 1, threadAmount, length);
    mergeSortChunk(channel, bucket + chunkStart, chunkLength);
    __syncthreads();
    while (range < threadAmount) {
        if (iterationControl % 2 == 0) {
            int firstChunkLength = getChunkLength(threadIndex, threadIndex + range, threadAmount, length);
            int secondChunkLength = getChunkLength(threadIndex + range, threadIndex + 2 * range, threadAmount, length);
            mergeChunk(channel, bucket + chunkStart, firstChunkLength, bucket + chunkStart + firstChunkLength, secondChunkLength);
            iterationControl /= 2;
        }
        range *= 2;
        __syncthreads();
    }
}

color getColorFromBucket(color* bucket, int bucketLength) {
    int redSum = 0, greenSum = 0, blueSum = 0;
    for (int i = 0; i < bucketLength; i++) {
        redSum += bucket[i].values[RED];
        greenSum += bucket[i].values[GREEN];
        blueSum += bucket[i].values[BLUE];
    }
    color newColor;
    newColor.values[RED] = redSum / bucketLength;
    newColor.values[GREEN] = greenSum / bucketLength;
    newColor.values[BLUE] = blueSum / bucketLength;
    return newColor;
}

color *OMPPaletteGeneration(png_bytep* image_pointers, int height, int width) {
    color* newPalette = (color*)malloc(sizeof(*newPalette) * PALETTE_SIZE);
    // All the buckets used to determine the palette. Number of buckets = number of colors in palette.
    color** pixelBuckets = (color**)malloc(sizeof(*pixelBuckets) * PALETTE_SIZE);
    // The first bucket is (for now) going to be width * height pixels long.
    pixelBuckets[0] = (color*)malloc(sizeof(*pixelBuckets[0]) * width * height);
    // Put every pixel in the first bucket.
    int k = 0;
    #pragma omp parallel for firstprivate(k)
    for (int i = 0; i < height; i++) {
        png_bytep row = image_pointers[i];
        for (int j = 0; j < width; j++) {
            png_bytep px = &(row[j * 4]);
            pixelBuckets[0][k].values[RED] = px[RED];
            pixelBuckets[0][k].values[GREEN] = px[GREEN];
            pixelBuckets[0][k++].values[BLUE] = px[BLUE];
        }
    }

    // Store the lenghts of each bucket and modify them as they change
    int bucketLengths[PALETTE_SIZE];
    bucketLengths[0] = width * height;
    for (int n = 0; n < PALETTE_SIZE - 1; n++) {
        //Iterate through every created bucket and find the one with the highest range of values in any channel.
        max_range mr_red, mr_blue, mr_green;
        mr_red.value = 0;
        mr_blue.value = 0;
        mr_green.value = 0;
        mr_red.bucket_index = 0;
        mr_blue.bucket_index = 0;
        mr_green.bucket_index = 0;
        #pragma omp parallel for
        for (int m = 0; m <= n; m++) {
            // Define max and min values of each channel. They will always be initialized on the first pass.
            int max_red, min_red, max_green, min_green, max_blue, min_blue;
            int isFirst = true;
            for (int i = 0; i < bucketLengths[m]; i++) {
                color px = pixelBuckets[m][i];
                if (isFirst) {
                        min_red = px.values[RED];
                        max_red = px.values[RED];
                        min_green = px.values[GREEN];
                        max_green = px.values[GREEN];
                        min_blue = px.values[BLUE];
                        max_blue = px.values[BLUE];
                        isFirst = false;
                }
                else {
                    if (px.values[RED] < min_red) {
                        min_red = px.values[RED];
                    }
                    else if (px.values[RED] > max_red) {
                        max_red = px.values[RED];
                    }
                    if (px.values[GREEN] < min_green) {
                        min_green = px.values[GREEN];
                    }
                    else if (px.values[GREEN] > max_green) {
                        max_green = px.values[GREEN];
                    }
                    if (px.values[BLUE] < min_blue) {
                        min_blue = px.values[BLUE];
                    }
                    else if (px.values[BLUE] > max_blue) {
                        max_blue = px.values[BLUE];
                    }
                }
            }
            #pragma omp critical
            {
            if (max_red - min_red > mr_red.value) {
                mr_red.value = max_red - min_red;
                mr_red.bucket_index = m;
            }
            }
            #pragma omp critical
            {
            if (max_blue - min_blue > mr_blue.value) {
                mr_blue.value = max_blue - min_blue;
                mr_blue.bucket_index = m;
            }
            }
            #pragma omp critical
            {
            if (max_green - min_green > mr_green.value) {
                mr_green.value = max_green - min_green;
                mr_green.bucket_index = m;
            }
            }
        }
        int chosenBucket = 0, chosenChannel;
        if (mr_red.value >= mr_green.value) {
            if (mr_red.value >= mr_blue.value) {
                chosenBucket = mr_red.bucket_index;
                chosenChannel = RED;
            }
            else {
                chosenBucket = mr_blue.bucket_index;
                chosenChannel = BLUE;
            }
        }
        else {
            if (mr_green.value >= mr_blue.value) {
                chosenBucket = mr_green.bucket_index;
                chosenChannel = GREEN;
            }
            else {
                chosenBucket = mr_blue.bucket_index;
                chosenChannel = BLUE;
            }
        }
        color* controlBucket = (color*)malloc(sizeof(color) * bucketLengths[chosenBucket]);
        memcpy(controlBucket, pixelBuckets[chosenBucket], bucketLengths[chosenBucket] * sizeof(color));
        sequentialSortBucket(chosenChannel, controlBucket, bucketLengths[chosenBucket]);
        
        dim3 numBlocks(blocks);
        dim3 threadsPerBlock(threads);
        color* gpuChosenBucket;
        hipMalloc(&gpuChosenBucket, sizeof(*gpuChosenBucket) * bucketLengths[chosenBucket]);
        hipMemcpy(gpuChosenBucket, pixelBuckets[chosenBucket], sizeof(*gpuChosenBucket) * bucketLengths[chosenBucket], hipMemcpyHostToDevice);
        //sortBucket<<<numBlocks, threadsPerBlock>>>(chosenChannel, gpuChosenBucket, bucketLengths[chosenBucket]);
        hipMemcpy(pixelBuckets[chosenBucket], gpuChosenBucket, sizeof(*gpuChosenBucket) * bucketLengths[chosenBucket], hipMemcpyDeviceToHost);
        hipFree(gpuChosenBucket);
        sequentialSortBucket(chosenChannel, pixelBuckets[chosenBucket], bucketLengths[chosenBucket]);
        int mid = bucketLengths[chosenBucket] / 2;
        pixelBuckets[n + 1] = (color*)malloc(sizeof(*pixelBuckets[n + 1]) * bucketLengths[chosenBucket] - mid);
        bucketLengths[n + 1] = bucketLengths[chosenBucket] - mid;
        for (int i = 0; i < bucketLengths[chosenBucket] - mid; i++) {
            pixelBuckets[n + 1][i] = pixelBuckets[chosenBucket][mid + i];
        }
        pixelBuckets[chosenBucket] = (color*)realloc(pixelBuckets[chosenBucket], mid * sizeof(*pixelBuckets[chosenBucket]));
        bucketLengths[chosenBucket] = mid;
    }
    for (int n = 0; n < PALETTE_SIZE; n++) {
        newPalette[n] = getColorFromBucket(pixelBuckets[n], bucketLengths[n]);
        free(pixelBuckets[n]);
    }
    free(pixelBuckets);
    return newPalette;
}

void applyPalette(png_bytep* row_pointers, color* palette, int height, int width) {
    for (int i = 0; i < height; i++) {
        png_bytep row = row_pointers[i];
        for (int j = 0; j < width; j++) {
            png_bytep px = row + 4 * j;
            // The most distant two colors can be from each other is ~443 units. Distance between colors is calculated with the following formula:
            // sqrt(red_dif^2 + green_dif^2 + blue_dif^2)
            // best_color_sim's distance is initialized to 444 (an impossible value), so as to make sure the struct is updated with a valid color from the palette on the first iteration
            color_similarity best_color_sim;
            best_color_sim.distance = 444;
            best_color_sim.color_index = -1;
            for (int k = 0; k < PALETTE_SIZE; k++) {
                color currentColor = palette[k];
                double currentDistance;
                currentDistance = sqrt(pow(currentColor.values[RED] - px[RED], 2) + pow(currentColor.values[GREEN] - px[GREEN], 2) + pow(currentColor.values[BLUE] - px[BLUE], 2));
                if (currentDistance < best_color_sim.distance) {
                    best_color_sim.color_index = k;
                    best_color_sim.distance = currentDistance;
                }
            }
            px[RED] = palette[best_color_sim.color_index].values[RED];
            px[GREEN] = palette[best_color_sim.color_index].values[GREEN];
            px[BLUE] = palette[best_color_sim.color_index].values[BLUE];
        }
    }
}

void writeFile(int width, int height, png_bytep* row_pointers, FILE* newImage) {
        png_structp png_write = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
        if (!png_write) abort();

        png_infop info_write = png_create_info_struct(png_write);
        if (!info_write) abort();

        if (setjmp(png_jmpbuf(png_write))) abort();

        png_init_io(png_write, newImage);

        png_set_IHDR(
            png_write,
            info_write,
            width, height,
            8,
            PNG_COLOR_TYPE_RGBA,
            PNG_INTERLACE_NONE,
            PNG_COMPRESSION_TYPE_DEFAULT,
            PNG_FILTER_TYPE_DEFAULT
        );
        png_write_info(png_write, info_write);

        if (!row_pointers) abort();

        png_write_image(png_write, row_pointers);
        png_write_end(png_write, NULL);
        png_destroy_write_struct(&png_write, &info_write);
}

void preprocessing(FILE* image, int* width_ptr, int* height_ptr,
                   png_structp png, png_bytep color_type,
                   png_bytep bit_depth, png_infop info) {
    png_init_io(png, image);

    png_read_info(png, info);

    *width_ptr = png_get_image_width(png, info);
    *height_ptr = png_get_image_height(png, info);
    *color_type = png_get_color_type(png, info);
    *bit_depth = png_get_bit_depth(png, info);
    if (*bit_depth == 16) {
        png_set_strip_16(png);
    }

    if (*color_type == PNG_COLOR_TYPE_PALETTE) {
        png_set_palette_to_rgb(png);
    }

    if (*color_type == PNG_COLOR_TYPE_GRAY && *bit_depth < 8) {
        png_set_expand_gray_1_2_4_to_8(png);
    }

    if (png_get_valid(png, info, PNG_INFO_tRNS)) {
        png_set_tRNS_to_alpha(png);
    }

    if (*color_type == PNG_COLOR_TYPE_RGB ||
        *color_type == PNG_COLOR_TYPE_GRAY ||
        *color_type == PNG_COLOR_TYPE_PALETTE) {
            png_set_filler(png, 0xFF, PNG_FILLER_AFTER);
        }

    if (*color_type == PNG_COLOR_TYPE_GRAY || *color_type == PNG_COLOR_TYPE_GRAY_ALPHA) {
        png_set_gray_to_rgb(png);
    }

    png_read_update_info(png, info);
    
}

void OMPProcessingLoop(char** files, int dirlen) {
    #pragma omp parallel for
    for (int i = 0; i < dirlen; i++) {
        png_structp png = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
        if (!png) {
            abort();
        }

        png_infop info = png_create_info_struct(png);
        if (!info) {
            abort();
        }

        if (setjmp(png_jmpbuf(png))) {
            abort();
        }
        FILE *image = fopen(files[i], "r");
        int height;
        int width;
        png_byte color_type;
        png_byte bit_depth;
        preprocessing(image, &width, &height, png, &color_type, &bit_depth, info);

        png_bytep *row_pointers = NULL;
        if (row_pointers) abort();
        row_pointers = (png_bytep*)malloc(sizeof(png_bytep) * height);
        for (int i = 0; i < height; i++) {
            row_pointers[i] = (png_bytep)malloc(png_get_rowbytes(png, info));
        }

        png_read_image(png, row_pointers);

        fclose(image);
        png_destroy_read_struct(&png, &info, NULL);

        color* palette = OMPPaletteGeneration(row_pointers, height, width);
        applyPalette(row_pointers, palette, height, width);

        // Write to a new image.
        char* newFileName = (char*)malloc(sizeof(char) * (strlen(files[i])));
        strcpy(newFileName, files[i]);
        strncpy(newFileName + 2, "edited", 6);
        FILE *newImage = fopen(newFileName, "wb");
        if (!newImage) abort();

        writeFile(width, height, row_pointers, newImage);

        for (int i = 0; i < height; i++) {
            free(row_pointers[i]);
        }

        free(row_pointers);
        fclose(newImage);
        free(palette);

    }
}

int main(int argc, char* argv[]) {
    if (argc > 1) {
        threads = atoi(argv[1]);
        if (argc > 2) {
            blocks = atoi(argv[2]);
            if (argc > 3) {
                PALETTE_SIZE = atoi(argv[3]);
            }
        }
    }
    char** files = (char**)malloc(0);
    DIR *dr = opendir("./images/");
    struct dirent *en;
    int dirlen = 0;
    size_t size;
    hipDeviceSetLimit(hipLimitStackSize, 8192);
    hipDeviceGetLimit(&size, hipLimitStackSize);
    hipDeviceSetLimit(hipLimitMallocHeapSize, 1280000000);
    printf("%d", size);

    if (dr) {
        while ((en = readdir(dr)) != NULL) {
            if (en->d_name[0] != '.') {
                files = (char**)realloc(files, (dirlen + 1) * sizeof(char*));
                files[dirlen] = (char*)malloc((strlen(en->d_name) + PATH_LEN) * sizeof(char));
                strcpy(files[dirlen], "./images/");
                strcpy(files[dirlen] + PATH_LEN, en->d_name);
                dirlen += 1;
            }
        }
        closedir(dr);
    }

    OMPProcessingLoop(files, dirlen);
}

